// std
#include <iostream>
#include <string>

#include "hip/hip_runtime_api.h"

int main(int argc, char** argv) {
    int deviceCount;
    checkCudaErrors(hipGetDeviceCount(&deviceCount));

    std::cout << deviceCount << " devices found" << std::endl;

    return 0;
}
